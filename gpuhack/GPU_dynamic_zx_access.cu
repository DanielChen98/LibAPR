#include "hip/hip_runtime.h"
//
// Created by cheesema on 09.03.18.
//
#include <algorithm>
#include <vector>
#include <array>
#include <iostream>
#include <cassert>
#include <limits>
#include <chrono>
#include <iomanip>

#include "data_structures/APR/APR.hpp"
#include "data_structures/APR/APRTreeIterator.hpp"
#include "data_structures/APR/ExtraParticleData.hpp"
#include "data_structures/Mesh/MeshData.hpp"
#include "io/TiffUtils.hpp"

#include "thrust/device_vector.h"
#include "thrust/tuple.h"
#include "thrust/copy.h"

#define X_MASK ((((uint64_t)1) << 2) - 1)
#define X_SHIFT 1




struct cmdLineOptions{
    std::string output = "output";
    std::string stats = "";
    std::string directory = "";
    std::string input = "";
};

bool command_option_exists(char **begin, char **end, const std::string &option) {
    return std::find(begin, end, option) != end;
}

char* get_command_option(char **begin, char **end, const std::string &option) {
    char ** itr = std::find(begin, end, option);
    if (itr != end && ++itr != end) {
        return *itr;
    }
    return 0;
}

cmdLineOptions read_command_line_options(int argc, char **argv) {
    cmdLineOptions result;

    if(argc == 1) {
        std::cerr << "Usage: \"Example_apr_neighbour_access -i input_apr_file -d directory\"" << std::endl;
        exit(1);
    }
    if(command_option_exists(argv, argv + argc, "-i")) {
        result.input = std::string(get_command_option(argv, argv + argc, "-i"));
    } else {
        std::cout << "Input file required" << std::endl;
        exit(2);
    }
    if(command_option_exists(argv, argv + argc, "-d")) {
        result.directory = std::string(get_command_option(argv, argv + argc, "-d"));
    }
    if(command_option_exists(argv, argv + argc, "-o")) {
        result.output = std::string(get_command_option(argv, argv + argc, "-o"));
    }

    return result;
}


void create_test_particles_surya(APR<uint16_t>& apr,APRIterator<uint16_t>& apr_iterator,ExtraParticleData<float> &test_particles,ExtraParticleData<uint16_t>& particles,std::vector<float>& stencil, const int stencil_size, const int stencil_half);








uint64_t encode_xzl(uint64_t x,uint64_t z,uint64 level){







}






////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv) {
    // Read provided APR file
    cmdLineOptions options = read_command_line_options(argc, argv);
    const int reps = 20;

    std::string fileName = options.directory + options.input;
    APR<uint16_t> apr;
    apr.read_apr(fileName);

    // Get dense representation of APR
    APRIterator<uint16_t> aprIt(apr);

    ///////////////////////////
    ///
    /// Sparse Data for GPU
    ///
    ///////////////////////////

    std::vector<std::tuple<std::size_t,std::size_t>> level_zx_index_start;//size = number of rows on all levels
    std::vector<std::uint16_t> y_explicit;y_explicit.reserve(aprIt.total_number_particles());//size = number of particles
    std::vector<std::uint16_t> particle_values;particle_values.reserve(aprIt.total_number_particles());//size = number of particles
    std::vector<std::size_t> level_offset(aprIt.level_max()+1,UINT64_MAX);//size = number of levels
    const int stencil_half = 2;
    const int stencil_size = 2*stencil_half+1;
    std::vector<std::float_t> stencil;		// the stencil on the host
    std::float_t stencil_value = 1;
    stencil.resize(pow(stencil_half*2 + 1,stencil_size),stencil_value);

    std::cout << stencil[0] << std::endl;


    std::size_t x = 0;
    std::size_t z = 0;

    std::size_t zx_counter = 0;
    std::size_t pcounter = 0;


    uint64_t bundle_xzl=0;


    for (int level = aprIt.level_min(); level <= aprIt.level_max(); ++level) {
        level_offset[level] = zx_counter;

        for (z = 0; z < aprIt.spatial_index_z_max(level); ++z) {
            for (x = 0; x < aprIt.spatial_index_x_max(level); ++x) {

                zx_counter++;
                if (aprIt.set_new_lzx(level, z, x) < UINT64_MAX) {
                    level_zx_index_start.emplace_back(std::make_tuple<std::size_t,std::size_t>(aprIt.global_index(),
                                                                                               aprIt.particles_zx_end(level,z,x))); //This stores the begining and end global index for each level_xz_row
                } else {
                    level_zx_index_start.emplace_back(std::make_tuple<std::size_t,std::size_t>((std::size_t)pcounter,(std::size_t) pcounter)); //This stores the begining and end global index for each level_
                }

                for (aprIt.set_new_lzx(level, z, x);
                     aprIt.global_index() < aprIt.particles_zx_end(level, z,
                                                                   x); aprIt.set_iterator_to_particle_next_particle()) {
                    y_explicit.emplace_back(aprIt.y());
                    particle_values.emplace_back(apr.particles_intensities[aprIt]);
                    pcounter++;

                }
            }

        }
    }


    ////////////////////
    ///
    /// Example of doing our level,z,x access using the GPU data structure
    ///
    /////////////////////
    auto start = std::chrono::high_resolution_clock::now();


    thrust::host_vector<thrust::tuple<std::size_t,std::size_t> > h_level_zx_index_start(level_zx_index_start.size());
    thrust::transform(level_zx_index_start.begin(), level_zx_index_start.end(),
                      h_level_zx_index_start.begin(),
                      [] ( const auto& _el ){
                          return thrust::make_tuple(std::get<0>(_el), std::get<1>(_el));
                      } );

    thrust::device_vector<thrust::tuple<std::size_t,std::size_t> > d_level_zx_index_start = h_level_zx_index_start;


    thrust::device_vector<std::float_t> d_stencil(stencil.begin(), stencil.end());		// device stencil
    thrust::device_vector<std::uint16_t> d_y_explicit(y_explicit.begin(), y_explicit.end());
    thrust::device_vector<std::uint16_t> d_particle_values(particle_values.begin(), particle_values.end());
    thrust::device_vector<std::uint16_t> d_test_access_data(d_particle_values.size(),0);

    thrust::device_vector<std::size_t> d_level_offset(level_offset.begin(),level_offset.end());


    std::size_t number_blocks = 8000;

    thrust::device_vector<std::uint16_t> d_x_end(number_blocks,0);
    std::uint16_t*   _x_end  =  thrust::raw_pointer_cast(d_x_end.data());

    thrust::device_vector<std::size_t> d_ind_end(number_blocks,0);
    std::size_t*   _ind_end  =  thrust::raw_pointer_cast(d_ind_end.data());

    std::size_t max_elements = 0;

    for (int level = aprIt.level_min(); level <= aprIt.level_max(); ++level) {
        auto xtimesy = aprIt.spatial_index_y_max(level);// + (stencil_size - 1);
        xtimesy *= aprIt.spatial_index_x_max(level);// + (stencil_size - 1);
        if(max_elements < xtimesy)
            max_elements = xtimesy;
    }
    thrust::device_vector<std::uint16_t> d_temp_vec(max_elements*stencil_size,0);

    const thrust::tuple<std::size_t,std::size_t>* levels =  thrust::raw_pointer_cast(d_level_zx_index_start.data());
    const std::uint16_t*             y_ex   =  thrust::raw_pointer_cast(d_y_explicit.data());
    const std::uint16_t*             pdata  =  thrust::raw_pointer_cast(d_particle_values.data());
    const std::size_t*             offsets= thrust::raw_pointer_cast(d_level_offset.data());
    std::uint16_t*                   tvec = thrust::raw_pointer_cast(d_temp_vec.data());
    std::uint16_t*                   expected = thrust::raw_pointer_cast(d_test_access_data.data());
    const std::float_t*		     stencil_pointer =  thrust::raw_pointer_cast(d_stencil.data());		// stencil pointer



    //////////////////////////
    ///
    /// Now check the data
    ///
    ////////////////////////////

    ExtraParticleData<float> utest_data(apr);
    apr.parameters.input_dir = options.directory;

    //create_test_particles_surya(apr,aprIt, utest_data,apr.particles_intensities,stencil, stencil_size, stencil_half);

    bool success = true;

    uint64_t c_fail= 0;



    for (uint64_t particle_number = 0; particle_number < apr.total_number_particles(); ++particle_number) {
        //This step is required for all loops to set the iterator by the particle number
        aprIt.set_iterator_to_particle_by_number(particle_number);


        if(utest_data.data[particle_number]!=test_access_data[particle_number]){
            success = false;

                //if(aprIt.level() == 6) {
//                    std::cout << particle_number << std::endl;
//                std::cout << aprIt.x() << " " << aprIt.y() << " " << aprIt.z() << " " << aprIt.level() << " expected: "
//                          << utest_data.data[particle_number] << ", received: " << test_access_data[particle_number]
//                          << std::endl;
            //}
            //break;
            c_fail++;
        }

        // std::cout << aprIt.x()<< " "  << aprIt.y()<< " "  << aprIt.z() << " "<< aprIt.level() << " expected: " << utest_data.data[particle_number] << ", received: " << test_access_data[particle_number] << "\n";

    }


    if(success){
        std::cout << "PASS" << std::endl;
    } else {
        std::cout << "FAIL " << c_fail << std::endl;
    }


}

__global__ void load_balance_xzl(const uint16_t level_,const thrust::tuple<std::size_t,std::size_t>* _line_offsets,std::uint16_t*  _xend,const std::size_t* _offsets,
                                 std::size_t   _max_x,std::size_t num_blocks,std::float_t parts_per_block,std::size_t parts_begin){

    int x_index = blockDim.x * blockIdx.x + threadIdx.x;
    int z_index = blockDim.y * blockIdx.y + threadIdx.y;

    if(x_index >= _max_x){
        return; // out of bounds
    }

    //printf("Hello from dim: %d block: %d, thread: %d  x index: %d z: %d \n",blockDim.x, blockIdx.x, threadIdx.x,x_index,(int) _z_index);

    auto level_zx_offset = _offsets[_level] + _max_x * _z_index + x_index;

    std::size_t parts_end = thrust::get<1>(_line_offsets[level_zx_offset]);

    std::size_t index_begin =  floor((thrust::get<0>(_line_offsets[level_zx_offset])-parts_begin)/parts_per_block);

    std::size_t index_end;

    if(parts_end==parts_begin){
        index_end=0;
    } else {
        index_end = floor((parts_end-parts_begin)/parts_per_block);
    }

    //need to add the loop
    if(index_begin!=index_end){


        for (int i = (index_begin+1); i <= index_end; ++i) {
            _xend[i]=x_index;

        }
    }


    if(x_index==(_max_x-1)){
        _ind_end[num_blocks-1] = parts_end;
        _xend[num_blocks-1] = (_max_x-1);

    }



}



void create_test_particles_surya(APR<uint16_t>& apr,APRIterator<uint16_t>& apr_iterator,ExtraParticleData<float> &test_particles,ExtraParticleData<uint16_t>& particles,std::vector<float>& stencil, const int stencil_size, const int stencil_half){

    for (uint64_t level_local = apr_iterator.level_max(); level_local >= apr_iterator.level_min(); --level_local) {


        MeshData<float> by_level_recon;
        by_level_recon.init(apr_iterator.spatial_index_y_max(level_local),apr_iterator.spatial_index_x_max(level_local),apr_iterator.spatial_index_z_max(level_local),0);

        uint64_t level = level_local;

        const int step_size = 1;

        uint64_t particle_number;

        for (particle_number = apr_iterator.particles_level_begin(level);
             particle_number < apr_iterator.particles_level_end(level); ++particle_number) {
            //
            //  Parallel loop over level
            //
            apr_iterator.set_iterator_to_particle_by_number(particle_number);

            int dim1 = apr_iterator.y() ;
            int dim2 = apr_iterator.x() ;
            int dim3 = apr_iterator.z() ;

            float temp_int;
            //add to all the required rays

            temp_int = particles[apr_iterator];

            const int offset_max_dim1 = std::min((int) by_level_recon.y_num, (int) (dim1 + step_size));
            const int offset_max_dim2 = std::min((int) by_level_recon.x_num, (int) (dim2 + step_size));
            const int offset_max_dim3 = std::min((int) by_level_recon.z_num, (int) (dim3 + step_size));

            for (int64_t q = dim3; q < offset_max_dim3; ++q) {

                for (int64_t k = dim2; k < offset_max_dim2; ++k) {
                    for (int64_t i = dim1; i < offset_max_dim1; ++i) {
                        by_level_recon.mesh[i + (k) * by_level_recon.y_num + q * by_level_recon.y_num * by_level_recon.x_num] = temp_int;
                    }
                }
            }
        }


        int x = 0;
        int z = 0;


        for (z = 0; z < (apr.spatial_index_z_max(level)); ++z) {
            //lastly loop over particle locations and compute filter.
            for (x = 0; x < apr.spatial_index_x_max(level); ++x) {
                for (apr_iterator.set_new_lzx(level, z, x);
                     apr_iterator.global_index() < apr_iterator.particles_zx_end(level, z,
                                                                                 x); apr_iterator.set_iterator_to_particle_next_particle()) {
                    double neigh_sum = 0;
                    float counter = 0;

                    const int k = apr_iterator.y(); // offset to allow for boundary padding
                    const int i = x;

                    //test_particles[apr_iterator]=0;

                    for (int l = -stencil_half; l < stencil_half+1; ++l) {
                        for (int q = -stencil_half; q < stencil_half+1; ++q) {
                            for (int w = -stencil_half; w < stencil_half+1; ++w) {

                                if((k+w)>=0 & (k+w) < (apr.spatial_index_y_max(level))){
                                    if((i+q)>=0 & (i+q) < (apr.spatial_index_x_max(level))){
                                        if((z+l)>=0 & (z+l) < (apr.spatial_index_z_max(level))){
                                            neigh_sum += stencil[counter] * by_level_recon.at(k + w, i + q, z+l);
                                            //neigh_sum += by_level_recon.at(k + w, i + q, z+l);
                                            //if(l==1) {
                                            //  test_particles[apr_iterator] = by_level_recon.at(k, i , z+l);
                                            //}
                                        }
                                    }
                                }
                                counter++;
                            }
                        }
                    }

                    test_particles[apr_iterator] = std::round(neigh_sum/(pow(stencil_size,3)*1.0));
                    test_particles[apr_iterator] = 1;

                }
            }
        }

        //std::string image_file_name = apr.parameters.input_dir + std::to_string(level_local) + "_by_level.tif";
       // TiffUtils::saveMeshAsTiff(image_file_name, by_level_recon);
    }

}



