#include "hip/hip_runtime.h"
//
// Created by cheesema on 09.03.18.
//
#include <algorithm>
#include <vector>
#include <array>
#include <iostream>
#include <cassert>
#include <limits>
#include <chrono>
#include <iomanip>

#include "data_structures/APR/APR.hpp"
#include "data_structures/APR/APRTreeIterator.hpp"
#include "data_structures/APR/ExtraParticleData.hpp"
#include "data_structures/Mesh/MeshData.hpp"
#include "io/TiffUtils.hpp"

#include "thrust/device_vector.h"
#include "thrust/tuple.h"
#include "thrust/copy.h"
#include "../src/misc/APRTimer.hpp"
#include "../src/data_structures/APR/ExtraParticleData.hpp"

#define KEY_EMPTY_MASK ((((uint64_t)1) << 1) - 1) << 0 //first bit stores if the row is empty or not can be used to avoid computations and accessed using &key
#define KEY_EMPTY_SHIFT 0

#define KEY_X_MASK ((((uint64_t)1) << 16) - 1) << 1
#define KEY_X_SHIFT 1

#define KEY_Z_MASK ((((uint64_t)1) << 16) - 1) << 17
#define KEY_Z_SHIFT 17

#define KEY_LEVEL_MASK ((((uint64_t)1) << 8) - 1) << 33
#define KEY_LEVEL_SHIFT 33



uint64_t encode_xzl(uint16_t x,uint16_t z,uint8_t level,bool nonzero){

    uint64_t raw_key=0;

    raw_key |= ((uint64_t)x << KEY_X_SHIFT);
    raw_key |= ((uint64_t)z << KEY_Z_SHIFT);
    raw_key |= ((uint64_t)level << KEY_LEVEL_SHIFT);

    if(nonzero){
        raw_key |= (1 << KEY_EMPTY_SHIFT);
    } else {
        raw_key |= (0 << KEY_EMPTY_SHIFT);
    }


    uint64_t output_x = (raw_key & KEY_X_MASK) >> KEY_X_SHIFT;
    uint64_t output_z = (raw_key & KEY_Z_MASK) >> KEY_Z_SHIFT;
    uint64_t output_level = (raw_key & KEY_LEVEL_MASK) >> KEY_LEVEL_SHIFT;
    uint64_t output_nz = (raw_key & KEY_EMPTY_MASK) >> KEY_EMPTY_SHIFT;

    uint64_t short_nz = raw_key&1;

    return raw_key;

}

bool decode_xzl(std::uint64_t raw_key,uint16_t& output_x,uint16_t& output_z,uint8_t& output_level){


    output_x = (raw_key & KEY_X_MASK) >> KEY_X_SHIFT;
    output_z = (raw_key & KEY_Z_MASK) >> KEY_Z_SHIFT;
    output_level = (raw_key & KEY_LEVEL_MASK) >> KEY_LEVEL_SHIFT;


    return raw_key&1;

}



struct cmdLineOptions{
    std::string output = "output";
    std::string stats = "";
    std::string directory = "";
    std::string input = "";
};

bool command_option_exists(char **begin, char **end, const std::string &option) {
    return std::find(begin, end, option) != end;
}

char* get_command_option(char **begin, char **end, const std::string &option) {
    char ** itr = std::find(begin, end, option);
    if (itr != end && ++itr != end) {
        return *itr;
    }
    return 0;
}

cmdLineOptions read_command_line_options(int argc, char **argv) {
    cmdLineOptions result;

    if(argc == 1) {
        std::cerr << "Usage: \"Example_apr_neighbour_access -i input_apr_file -d directory\"" << std::endl;
        exit(1);
    }
    if(command_option_exists(argv, argv + argc, "-i")) {
        result.input = std::string(get_command_option(argv, argv + argc, "-i"));
    } else {
        std::cout << "Input file required" << std::endl;
        exit(2);
    }
    if(command_option_exists(argv, argv + argc, "-d")) {
        result.directory = std::string(get_command_option(argv, argv + argc, "-d"));
    }
    if(command_option_exists(argv, argv + argc, "-o")) {
        result.output = std::string(get_command_option(argv, argv + argc, "-o"));
    }

    return result;
}


void create_test_particles_surya(APR<uint16_t>& apr,APRIterator<uint16_t>& apr_iterator,ExtraParticleData<float> &test_particles,ExtraParticleData<uint16_t>& particles,std::vector<float>& stencil, const int stencil_size,
                                 const int stencil_half);


__global__ void load_balance_xzl(const thrust::tuple<std::size_t,std::size_t>* row_info,std::size_t*  _chunk_index_end,
                                 std::size_t total_number_chunks,std::float_t parts_per_block,std::size_t total_number_rows);

__global__ void test_dynamic_balance(const thrust::tuple<std::size_t,std::size_t>* row_info,std::size_t*  _chunk_index_end,
                                     std::size_t total_number_chunks,const std::uint16_t* particle_y,std::uint16_t* particle_data_output);

__global__ void test_dynamic_balance_XZYL(const thrust::tuple<std::size_t,std::size_t>* row_info,std::size_t*  _chunk_index_end,
                                          std::size_t total_number_chunks,const std::uint16_t* particle_y,std::uint16_t* particle_data_output);


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv) {
    // Read provided APR file
    cmdLineOptions options = read_command_line_options(argc, argv);
    const int reps = 100;

    std::string fileName = options.directory + options.input;
    APR<uint16_t> apr;
    apr.read_apr(fileName);

    // Get dense representation of APR
    APRIterator<uint16_t> aprIt(apr);

#ifdef APR_USE_CUDA
    std::cout << "hello" << std::endl;
#endif



    ///////////////////////////
    ///
    /// Sparse Data for GPU
    ///
    ///////////////////////////

    std::vector<std::tuple<std::size_t,std::size_t>> level_zx_index_start;//size = number of rows on all levels
    std::vector<std::uint16_t> y_explicit;y_explicit.reserve(aprIt.total_number_particles());//size = number of particles
    std::vector<std::uint16_t> particle_values;particle_values.reserve(aprIt.total_number_particles());//size = number of particles
    std::vector<std::size_t> level_offset(aprIt.level_max()+1,UINT64_MAX);//size = number of levels


    std::size_t x = 0;
    std::size_t z = 0;

    std::size_t zx_counter = 0;
    std::size_t pcounter = 0;


    uint64_t bundle_xzl=0;

    APRTimer timer;
    timer.verbose_flag = true;

    timer.start_timer("initialize structure");

    for (int level = aprIt.level_min(); level <= aprIt.level_max(); ++level) {
        level_offset[level] = zx_counter;

        for (z = 0; z < aprIt.spatial_index_z_max(level); ++z) {
            for (x = 0; x < aprIt.spatial_index_x_max(level); ++x) {

                zx_counter++;
                uint64_t key;
                if (aprIt.set_new_lzx(level, z, x) < UINT64_MAX) {

                     key = encode_xzl(x,z,level,1);
                    level_zx_index_start.emplace_back(std::make_tuple<std::size_t,std::size_t>((std::size_t)key,
                                                                                               (std::size_t)aprIt.particles_zx_end(level,z,x))); //This stores the begining and end global index for each level_xz_row
                } else {
                     key = encode_xzl(x,z,level,0);
                    level_zx_index_start.emplace_back(std::make_tuple<std::size_t,std::size_t>((std::size_t)key,(std::size_t) pcounter)); //This stores the begining and end global index for each level_
                }


                for (aprIt.set_new_lzx(level, z, x);
                     aprIt.global_index() < aprIt.particles_zx_end(level, z,
                                                                   x); aprIt.set_iterator_to_particle_next_particle()) {
                    y_explicit.emplace_back(aprIt.y());
                    particle_values.emplace_back(apr.particles_intensities[aprIt]);
                    pcounter++;

                }
            }

        }
    }

    timer.stop_timer();



    ////////////////////
    ///
    /// Example of doing our level,z,x access using the GPU data structure
    ///
    /////////////////////
    timer.start_timer("transfer structures to GPU");


    uint64_t total_number_rows = level_zx_index_start.size();

    thrust::host_vector<thrust::tuple<std::size_t,std::size_t> > h_level_zx_index_start(level_zx_index_start.size());
    thrust::transform(level_zx_index_start.begin(), level_zx_index_start.end(),
                      h_level_zx_index_start.begin(),
                      [] ( const auto& _el ){
                          return thrust::make_tuple(std::get<0>(_el), std::get<1>(_el));
                      } );

    thrust::device_vector<thrust::tuple<std::size_t,std::size_t> > d_level_zx_index_start = h_level_zx_index_start;


    thrust::device_vector<std::uint16_t> d_y_explicit(y_explicit.begin(), y_explicit.end()); //y-coordinates
    thrust::device_vector<std::uint16_t> d_particle_values(particle_values.begin(), particle_values.end()); //particle values


    thrust::device_vector<std::size_t> d_level_offset(level_offset.begin(),level_offset.end()); //cumsum of number of rows in lower levels

    /*
     * Dynamic load balancing of the APR data-structure variables
     *
     */

    std::size_t max_number_chunks = 8191;
    thrust::device_vector<std::size_t> d_ind_end(max_number_chunks,0);
    std::size_t*   chunk_index_end  =  thrust::raw_pointer_cast(d_ind_end.data());

    const thrust::tuple<std::size_t,std::size_t>* row_info =  thrust::raw_pointer_cast(d_level_zx_index_start.data());
    const std::uint16_t*             particle_y   =  thrust::raw_pointer_cast(d_y_explicit.data());
    const std::uint16_t*             pdata  =  thrust::raw_pointer_cast(d_particle_values.data());
    const std::size_t*             offsets= thrust::raw_pointer_cast(d_level_offset.data());


    ExtraParticleData<uint16_t> iteration_check_particles(apr);
    iteration_check_particles.init_gpu(apr.total_number_particles());


    timer.stop_timer();

    /*
     * Dynamic load balancing of the APR data-structure variables
     *
     */

    timer.start_timer("load balancing");

    std::cout << "Total number of rows: " << total_number_rows << std::endl;

    std::size_t total_number_particles = apr.total_number_particles();

    //Figuring out how many particles per chunk are required
    std::size_t max_particles_per_row = apr.orginal_dimensions(0); //maximum number of particles in a row
    std::size_t parts_per_chunk = std::max((std::size_t)(max_particles_per_row+1),(std::size_t) floor(total_number_particles/max_number_chunks)); // to gurantee every chunk stradles across more then one row, the minimum particle chunk needs ot be larger then the largest possible number of particles in a row

    std::size_t actual_number_chunks = total_number_particles/parts_per_chunk + 1; // actual number of chunks realized based on the constraints on the total number of particles and maximum row

    dim3 threads(32);
    dim3 blocks((total_number_rows + threads.x - 1)/threads.x);

    std::cout << "Particles per chunk: " << parts_per_chunk << " Total number of chunks: " << actual_number_chunks << std::endl;

    load_balance_xzl<<<blocks,threads>>>(row_info,chunk_index_end,actual_number_chunks,parts_per_chunk,total_number_rows);
    hipDeviceSynchronize();

    timer.stop_timer();


    /*
     *  Now launch the kernels across all the chunks determiend by the load balancing
     *
     */


    int number_reps = 40;


    timer.start_timer("iterate over all particles");

    dim3 threads_dyn(32);
    dim3 blocks_dyn((actual_number_chunks + threads_dyn.x - 1)/threads_dyn.x);

    for (int rep = 0; rep < number_reps; ++rep) {

        test_dynamic_balance << < blocks_dyn, threads_dyn >> >
                                              (row_info, chunk_index_end, actual_number_chunks, particle_y, iteration_check_particles.gpu_pointer);
        hipDeviceSynchronize();
    }


    timer.stop_timer();

    float gpu_iterate_time = timer.timings.back();




    /*
     *  Off-load the particle data from the GPU
     *
     */

    timer.start_timer("output transfer from GPU");

    iteration_check_particles.copy_data_to_host();

    timer.stop_timer();

    /*
    *  Test the x,y,z,level information is correct
    *
    */


    ExtraParticleData<uint16_t> spatial_info_test(apr);
    spatial_info_test.init_gpu(apr.total_number_particles());



    timer.start_timer("summing the sptial informatino for each partilce on the GPU");
    for (int rep = 0; rep < number_reps; ++rep) {

        test_dynamic_balance_XZYL << < blocks_dyn, threads_dyn >> >
                                                   (row_info, chunk_index_end, actual_number_chunks, particle_y, spatial_info_test.gpu_pointer);

        hipDeviceSynchronize();
    }

    timer.stop_timer();

    float gpu_iterate_time_si = timer.timings.back();

    spatial_info_test.copy_data_to_host();



    /*
    *  Performance comparison with CPU
    *
    */

    ExtraParticleData<uint16_t> test_cpu(apr);

    timer.start_timer("Performance comparison on CPU serial");
    for (int rep = 0; rep < number_reps; ++rep) {
        for (uint64_t particle_number = 0; particle_number < apr.total_number_particles(); ++particle_number) {
            //This step is required for all loops to set the iterator by the particle number
            aprIt.set_iterator_to_particle_by_number(particle_number);

            test_cpu[aprIt] += 1;

        }
    }

    timer.stop_timer();

    float cpu_iterate_time = timer.timings.back();



    timer.start_timer("Performance comparison on CPU access sum"); //not working
    for (int rep = 0; rep < number_reps; ++rep) {

#pragma omp parallel for schedule(static) private(particle_number) firstprivate(aprIt)
        for (uint64_t particle_number = 0; particle_number < apr.total_number_particles(); ++particle_number) {
            //This step is required for all loops to set the iterator by the particle number
            aprIt.set_iterator_to_particle_by_number(particle_number);

            test_cpu[aprIt] = aprIt.x() + aprIt.y() + aprIt.z() + aprIt.level();

        }
    }

    timer.stop_timer();

    float cpu_iterate_time_si = timer.timings.back();

    std::cout << "SPEEDUP GPU vs. CPU iterate= " << cpu_iterate_time/gpu_iterate_time << std::endl;
    std::cout << "SPEEDUP GPU vs. CPU iterate (Spatial Info)= " << cpu_iterate_time_si/gpu_iterate_time_si << std::endl;

    //////////////////////////
    ///
    /// Now check the data
    ///
    ////////////////////////////


    bool success = true;

    uint64_t c_fail= 0;
    uint64_t c_pass= 0;

    for (uint64_t particle_number = 0; particle_number < apr.total_number_particles(); ++particle_number) {
        //This step is required for all loops to set the iterator by the particle number
        aprIt.set_iterator_to_particle_by_number(particle_number);
        if(iteration_check_particles[aprIt]==number_reps){
            c_pass++;
        } else {
            c_fail++;
            success = false;
            //std::cout << test_access_data[particle_number] << " Level: " < aprIt.level() << std::endl;
        }
    }

    if(success){
        std::cout << "Iteration Check, PASS" << std::endl;
    } else {
        std::cout << "Iteration Check, FAIL Total: " << c_fail << " Pass Total:  " << c_pass << std::endl;
    }


    /*
     *  Check the spatial data, by comparing x+y+z+level for every particle
     *
     */

    c_pass = 0;
    c_fail = 0;
    success=true;


    for (uint64_t particle_number = 0; particle_number < apr.total_number_particles(); ++particle_number) {
        //This step is required for all loops to set the iterator by the particle number
        aprIt.set_iterator_to_particle_by_number(particle_number);
        if(spatial_info_test[aprIt]==(aprIt.x() + aprIt.y() + aprIt.z() + aprIt.level())){
            c_pass++;
        } else {
            c_fail++;
            success = false;
            //std::cout << test_access_data[particle_number] << " Level: " < aprIt.level() << std::endl;
        }
    }

    if(success){
        std::cout << "Spatial information Check, PASS" << std::endl;
    } else {
        std::cout << "Spatial information Check, FAIL Total: " << c_fail << " Pass Total:  " << c_pass << std::endl;
    }

}

__global__ void test_dynamic_balance(const thrust::tuple<std::size_t,std::size_t>* row_info,std::size_t*  _chunk_index_end,
                                     std::size_t total_number_chunks,const std::uint16_t* particle_y,std::uint16_t* particle_data_output){

    int chunk_index = blockDim.x * blockIdx.x + threadIdx.x; // the input to each kernel is its chunk index for which it should iterate over

    if(chunk_index >= total_number_chunks){
        return; //out of bounds
    }

    //load in the begin and end row indexs
    std::size_t row_begin;
    std::size_t row_end;

    if(chunk_index==0){
        row_begin = 0;
    } else {
        row_begin = _chunk_index_end[chunk_index-1] + 1; //This chunk starts the row after the last one finished.
    }

    row_end = _chunk_index_end[chunk_index];

    std::size_t particle_global_index_begin;
    std::size_t particle_global_index_end;

    std::size_t current_row_key;

    for (std::size_t current_row = row_begin; current_row <= row_end; ++current_row) {
        current_row_key = thrust::get<0>(row_info[current_row]);
        if(current_row_key&1) { //checks if there any particles in the row

            particle_global_index_end = thrust::get<1>(row_info[current_row]);

            if (current_row == 0) {
                particle_global_index_begin = 0;
            } else {
                particle_global_index_begin = thrust::get<1>(row_info[current_row-1]);
            }

            //loop over the particles in the row
            for (std::size_t particle_global_index = particle_global_index_begin; particle_global_index < particle_global_index_end; ++particle_global_index) {

                particle_data_output[particle_global_index]+=1;
            }
        }
    }


}

__global__ void test_dynamic_balance_XZYL(const thrust::tuple<std::size_t,std::size_t>* row_info,std::size_t*  _chunk_index_end,
                                     std::size_t total_number_chunks,const std::uint16_t* particle_y,std::uint16_t* particle_data_output){

    int chunk_index = blockDim.x * blockIdx.x + threadIdx.x; // the input to each kernel is its chunk index for which it should iterate over

    if(chunk_index >= total_number_chunks){
        return; //out of bounds
    }

    //load in the begin and end row indexs
    std::size_t row_begin;
    std::size_t row_end;

    if(chunk_index==0){
        row_begin = 0;
    } else {
        row_begin = _chunk_index_end[chunk_index-1] + 1; //This chunk starts the row after the last one finished.
    }

    row_end = _chunk_index_end[chunk_index];

    std::size_t particle_global_index_begin;
    std::size_t particle_global_index_end;

    std::size_t current_row_key;

    for (std::size_t current_row = row_begin; current_row <= row_end; ++current_row) {
        current_row_key = thrust::get<0>(row_info[current_row]);
        if(current_row_key&1) { //checks if there any particles in the row

            particle_global_index_end = thrust::get<1>(row_info[current_row]);

            if (current_row == 0) {
                particle_global_index_begin = 0;
            } else {
                particle_global_index_begin = thrust::get<1>(row_info[current_row-1]);
            }

            std::uint16_t x;
            std::uint16_t z;
            std::uint8_t level;

            //decode the key
            x = (current_row_key & KEY_X_MASK) >> KEY_X_SHIFT;
            z = (current_row_key & KEY_Z_MASK) >> KEY_Z_SHIFT;
            level = (current_row_key & KEY_LEVEL_MASK) >> KEY_LEVEL_SHIFT;

            //loop over the particles in the row
            for (std::size_t particle_global_index = particle_global_index_begin; particle_global_index < particle_global_index_end; ++particle_global_index) {
                uint16_t current_y = particle_y[particle_global_index];
                particle_data_output[particle_global_index]=current_y+x+z+level;
            }

        }

    }


}


__global__ void load_balance_xzl(const thrust::tuple<std::size_t,std::size_t>* row_info,std::size_t*  _chunk_index_end,
                                 std::size_t total_number_chunks,std::float_t parts_per_block,std::size_t total_number_rows){

    int row_index = blockDim.x * blockIdx.x + threadIdx.x;

    if(row_index>=total_number_rows){
        return;
    }

//    std::size_t key= thrust::get<0>(row_info[row_index]);
//
//    if(!key&1){
//        return; //empty row
//    }

    std::size_t index_end = thrust::get<1>(row_info[row_index]);
    std::size_t index_begin;

    if(row_index > 0){
        index_begin = thrust::get<1>(row_info[row_index-1]);
    } else {
        index_begin =0;
    }

    std::size_t chunk_start = floor(index_begin/parts_per_block);
    std::size_t chunk_end =  floor(index_end/parts_per_block);

    if(chunk_start!=chunk_end){
        _chunk_index_end[chunk_end]=row_index;
    }

    if(row_index == (total_number_rows-1)){
        _chunk_index_end[total_number_chunks-1]=total_number_rows-1;
    }



}



void create_test_particles_surya(APR<uint16_t>& apr,APRIterator<uint16_t>& apr_iterator,ExtraParticleData<float> &test_particles,ExtraParticleData<uint16_t>& particles,std::vector<float>& stencil, const int stencil_size, const int stencil_half){

    for (uint64_t level_local = apr_iterator.level_max(); level_local >= apr_iterator.level_min(); --level_local) {


        MeshData<float> by_level_recon;
        by_level_recon.init(apr_iterator.spatial_index_y_max(level_local),apr_iterator.spatial_index_x_max(level_local),apr_iterator.spatial_index_z_max(level_local),0);

        uint64_t level = level_local;

        const int step_size = 1;

        uint64_t particle_number;

        for (particle_number = apr_iterator.particles_level_begin(level);
             particle_number < apr_iterator.particles_level_end(level); ++particle_number) {
            //
            //  Parallel loop over level
            //
            apr_iterator.set_iterator_to_particle_by_number(particle_number);

            int dim1 = apr_iterator.y() ;
            int dim2 = apr_iterator.x() ;
            int dim3 = apr_iterator.z() ;

            float temp_int;
            //add to all the required rays

            temp_int = particles[apr_iterator];

            const int offset_max_dim1 = std::min((int) by_level_recon.y_num, (int) (dim1 + step_size));
            const int offset_max_dim2 = std::min((int) by_level_recon.x_num, (int) (dim2 + step_size));
            const int offset_max_dim3 = std::min((int) by_level_recon.z_num, (int) (dim3 + step_size));

            for (int64_t q = dim3; q < offset_max_dim3; ++q) {

                for (int64_t k = dim2; k < offset_max_dim2; ++k) {
                    for (int64_t i = dim1; i < offset_max_dim1; ++i) {
                        by_level_recon.mesh[i + (k) * by_level_recon.y_num + q * by_level_recon.y_num * by_level_recon.x_num] = temp_int;
                    }
                }
            }
        }


        int x = 0;
        int z = 0;


        for (z = 0; z < (apr.spatial_index_z_max(level)); ++z) {
            //lastly loop over particle locations and compute filter.
            for (x = 0; x < apr.spatial_index_x_max(level); ++x) {
                for (apr_iterator.set_new_lzx(level, z, x);
                     apr_iterator.global_index() < apr_iterator.particles_zx_end(level, z,
                                                                                 x); apr_iterator.set_iterator_to_particle_next_particle()) {
                    double neigh_sum = 0;
                    float counter = 0;

                    const int k = apr_iterator.y(); // offset to allow for boundary padding
                    const int i = x;

                    //test_particles[apr_iterator]=0;

                    for (int l = -stencil_half; l < stencil_half+1; ++l) {
                        for (int q = -stencil_half; q < stencil_half+1; ++q) {
                            for (int w = -stencil_half; w < stencil_half+1; ++w) {

                                if((k+w)>=0 & (k+w) < (apr.spatial_index_y_max(level))){
                                    if((i+q)>=0 & (i+q) < (apr.spatial_index_x_max(level))){
                                        if((z+l)>=0 & (z+l) < (apr.spatial_index_z_max(level))){
                                            neigh_sum += stencil[counter] * by_level_recon.at(k + w, i + q, z+l);
                                            //neigh_sum += by_level_recon.at(k + w, i + q, z+l);
                                            //if(l==1) {
                                            //  test_particles[apr_iterator] = by_level_recon.at(k, i , z+l);
                                            //}
                                        }
                                    }
                                }
                                counter++;
                            }
                        }
                    }

                    test_particles[apr_iterator] = std::round(neigh_sum/(pow(stencil_size,3)*1.0));
                    test_particles[apr_iterator] = 1;

                }
            }
        }

        //std::string image_file_name = apr.parameters.input_dir + std::to_string(level_local) + "_by_level.tif";
       // TiffUtils::saveMeshAsTiff(image_file_name, by_level_recon);
    }

}



